#include<stdio.h>
#include<hip/hip_runtime.h>
#include<iostream>
#include<fstream>
#include<chrono>


using namespace std;

__global__ void serialReduction(int *d_array, int numberOfElements)
{
    int sum = 0;
    for(int i=0;i<numberOfElements;i++)
    {
        sum = sum + d_array[i];
    }
    printf("%d",sum);
}



void serialReduceHost(int *h_array,int *d_array, int numberOfElements)
{
    
    serialReduction<<<1,1>>>(d_array,numberOfElements);
    hipDeviceSynchronize();
    fflush(stdout);

}
__global__ void parallelReduction(int *d_array , int numberOfElements, int elementsPerThread,int numberOfThreadsPerBlock,int numberOfBlocks,int *d_global)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x ;
    index = index * elementsPerThread;

    if(index>numberOfElements)
    {
        return;
    }

    int sum = 0;

    for(int i=index;i<index+elementsPerThread;i++)
    {
        sum = sum + d_array[i];
    }

    extern __shared__ int d_blockMemmory[];
    d_blockMemmory[threadIdx.x] = sum;
    sum =0;
    __syncthreads();


    if(threadIdx.x == 0)
    {
        for(int i =0; i<numberOfThreadsPerBlock;i++)
        {
            sum = sum+ d_blockMemmory[i];
        }
        d_global[blockIdx.x] = sum;
    }
}
void parallelReduceHost(int *h_array ,int *d_array ,int numberOfElements,int elementsPerThread , int numberOfThreadsPerBlock , int numberOfBlocks)
{
    int *d_global;
    hipMalloc(&d_global, sizeof(int)*numberOfBlocks);


    parallelReduction<<<numberOfBlocks,numberOfThreadsPerBlock,numberOfThreadsPerBlock*sizeof(int)>>> (d_array,numberOfElements,elementsPerThread,numberOfThreadsPerBlock,numberOfBlocks,d_global);

    int *h_global = new int[numberOfBlocks];
    hipMemcpy(h_global,d_global,sizeof(int)*numberOfBlocks,hipMemcpyDeviceToHost);
    int sum=0;

    for(int i=0;i<numberOfBlocks;i++)

    {
        sum = sum + h_global[i];
    }

    printf("\n%d",sum);
}
void serialReduceCpu(int *d_array , int numberOfElements)
{
    int sum =0;
    for(int i=0;i<numberOfElements;i++)
    {
        sum = sum+d_array[i];
    }
    cout<<"\n"<<sum;
}
int main()
{
   
    int numberOfElements;
    ifstream inFile;
    inFile.open("random");
    int x;
    int i=0;
    inFile >>x ;
    numberOfElements = x;
   int *h_array = new int[numberOfElements];
    while(inFile >> x)
    {
        h_array[i] = x;
        i++;
    }
    int *d_array;
    hipMalloc(&d_array , sizeof(int)*numberOfElements);
    hipMemcpy(d_array, h_array , sizeof(int)*numberOfElements, hipMemcpyHostToDevice);
    //serialReduceHost(h_array, d_array ,numberOfElements);
    int elementsPerThread, numberOfBlocks , numberOfThreadsPerBlock;
   elementsPerThread = 256 ;
   numberOfThreadsPerBlock = 64;
   numberOfBlocks =6400;
    parallelReduceHost(h_array,d_array,numberOfElements,elementsPerThread,numberOfThreadsPerBlock ,numberOfBlocks);
    serialReduceCpu(h_array , numberOfElements);    


}   




//    freopen("random", "r", stdin);