#include<stdio.h>
#include<hip/hip_runtime.h>
#include<iostream>
#include<fstream>
#include<chrono>


using namespace std;
__global__ void parallelReduction(int *d_array , int numberOfElements, int elementsPerThread,int numberOfThreadsPerBlock,int numberOfBlocks,int *d_global)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x ;
    int sum = 0;

    int j=0;
    for(int i=index;i<numberOfElements;i = i+(numberOfBlocks*numberOfThreadsPerBlock))
    {
        sum = sum + d_array[i];
        j++;
    }
    extern __shared__ int d_blockMemmory[];
    d_blockMemmory[threadIdx.x] = sum;
    sum =0;
    __syncthreads();


    if(threadIdx.x == 0)
    {
        for(int i =0; i<numberOfThreadsPerBlock;i++)
        {
            sum = sum+ d_blockMemmory[i];
        }
        d_global[blockIdx.x] = sum;
    }
}
void parallelReduceHost(int *h_array ,int *d_array ,int numberOfElements,int elementsPerThread , int numberOfThreadsPerBlock , int numberOfBlocks)
{
    int *d_global;
    hipMalloc(&d_global, sizeof(int)*numberOfBlocks);


    parallelReduction<<<numberOfBlocks,numberOfThreadsPerBlock,numberOfThreadsPerBlock*sizeof(int)>>> (d_array,numberOfElements,elementsPerThread,numberOfThreadsPerBlock,numberOfBlocks,d_global);

    int *h_global = new int[numberOfBlocks];
    hipMemcpy(h_global,d_global,sizeof(int)*numberOfBlocks,hipMemcpyDeviceToHost);
    int sum=0;

    for(int i=0;i<numberOfBlocks;i++)

    {
        sum = sum + h_global[i];
    }

    printf("\n%d",sum);
}
int main()
{
   
    int numberOfElements;
    ifstream inFile;
    inFile.open("random");
    int x;
    int i=0;
    inFile >>x ;
    numberOfElements = x;
   int *h_array = new int[numberOfElements];
    while(inFile >> x)
    {
        h_array[i] = x;
        i++;
    }
    int *d_array;
    hipMalloc(&d_array , sizeof(int)*numberOfElements);
    hipMemcpy(d_array, h_array , sizeof(int)*numberOfElements, hipMemcpyHostToDevice);
    //serialReduceHost(h_array, d_array ,numberOfElements);
    int elementsPerThread, numberOfBlocks , numberOfThreadsPerBlock;
   elementsPerThread = 0 ;
   numberOfThreadsPerBlock = 64;
   numberOfBlocks =2560;
    parallelReduceHost(h_array,d_array,numberOfElements,elementsPerThread,numberOfThreadsPerBlock ,numberOfBlocks);  


}   
