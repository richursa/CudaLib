#include "hip/hip_runtime.h"
#include<iostream>
#include "random.h"
using namespace std;
__global__ void hillisSteeleScanDevice(int *d_array , int numberOfElements, int *d_tmpArray,int moveIndex)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    if(index > numberOfElements)
    {
        return;
    }
    if(index - moveIndex >=0)
    {
        d_tmpArray[index] = d_tmpArray[index] +d_array[index - moveIndex];
    }
    
}
void hillisSteeleScanHost(int *h_array,int numberOfElements)
{
    int *d_array;
    hipMalloc(&d_array,sizeof(int)*numberOfElements);
    hipMemcpy(d_array,h_array,sizeof(int)*numberOfElements,hipMemcpyHostToDevice);
    int *d_tmpArray;
    hipMalloc(&d_tmpArray,sizeof(int)*numberOfElements);
    for(int j=1;j<numberOfElements;j= j*2)
    {   hipMemcpy(d_tmpArray,d_array,sizeof(int)*numberOfElements,hipMemcpyDeviceToDevice);
        hillisSteeleScanDevice<<<1600,500>>>(d_array,numberOfElements,d_tmpArray, j);
        hipMemcpy(d_array,d_tmpArray,sizeof(int)*numberOfElements,hipMemcpyDeviceToDevice);
    }
    hipMemcpy(h_array,d_array ,sizeof(int)*numberOfElements,hipMemcpyDeviceToHost);
}

int main()
{
    cout<<"enter the number of numbers ";
    int numberOfElements;
    cin>>numberOfElements;
    int *h_array = new int[numberOfElements];
    class random a(h_array,numberOfElements);
    hillisSteeleScanHost(h_array,numberOfElements);
}
