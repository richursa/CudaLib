#include "hip/hip_runtime.h"
#include<iostream>
#include<stdio.h>
#include "random.h"
using namespace std;
__device__ int function(int value , int bit ,int bitset)
{
    if(bitset == 1 )
    {
        if((value & bit)  != 0)
        {
            return 1;
        }
        else 
            return 0;
    }
    else
    {
        if((value & bit) == 0)
        {
            return 1;
        }
        else 
        {
            return 0;
        }
    }
}
__global__ void predicateDevice(int *d_array , int *d_predicateArrry , int d_numberOfElements,int bit,int bitset)
{
    int index = threadIdx.x + blockIdx.x*blockDim.x;
    if(index < d_numberOfElements)
    {
    
           d_predicateArrry[index] = function(d_array[index],bit,bitset);
    }
}

__global__ void scatter(int *d_array , int *d_scanArray , int *d_predicateArrry,int * d_scatteredArray ,int d_numberOfElements,int offset)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if(index < d_numberOfElements)
    {
        if(d_predicateArrry[index] == 1)
        {
           // printf(" foundeed at index = %d val = %d\n",index,d_array[index]);
            d_scatteredArray[d_scanArray[index] - 1 +offset ] = d_array[index];
        
        }
    }
}
__global__ void hillisSteeleScanDevice(int *d_array , int numberOfElements, int *d_tmpArray,int moveIndex)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    if(index > numberOfElements)
    {
        return;
    }
    d_tmpArray[index] = d_array[index];
    if(index - moveIndex >=0)
    {
        
        d_tmpArray[index] = d_tmpArray[index] +d_array[index - moveIndex];
    }
}
int* hillisSteeleScanHost(int *d_scanArray,int numberOfElements)
{
    
    
    int *d_tmpArray;
    int *d_tmpArray1;
    hipMalloc(&d_tmpArray1,sizeof(int)*numberOfElements);
    hipMalloc(&d_tmpArray,sizeof(int)*numberOfElements);
    hipMemcpy(d_tmpArray1,d_scanArray,sizeof(int)*numberOfElements,hipMemcpyDeviceToDevice);
    int j,k=0;
    for(j=1;j<numberOfElements;j= j*2,k++)
    {
        if(k%2 == 0)
        {
            hillisSteeleScanDevice<<<1600,500>>>(d_tmpArray1,numberOfElements,d_tmpArray, j);
            hipDeviceSynchronize();
        }
        else
        {
            hillisSteeleScanDevice<<<1600,500>>>(d_tmpArray,numberOfElements,d_tmpArray1, j);
            hipDeviceSynchronize();
        }
    } 
    hipDeviceSynchronize();
    if(k%2 == 0)
    {
        
        return d_tmpArray1;
    }
    else
    {
        return d_tmpArray;
    }
}
__global__ void print(int *d_predicateArrry,int numberOfElements)
{
    
    for(int i=0;i<numberOfElements;i++)
    {
        printf("index = %d value = %d\n",i,d_predicateArrry[i]);
    }
}

int *compact(int *d_array,int numberOfElements,int bit)
{   
    int offset;
    int *d_predicateArrry;
    hipMalloc((void**)&d_predicateArrry,sizeof(int)*numberOfElements);
    predicateDevice<<<1600,500>>>(d_array,d_predicateArrry,numberOfElements,bit,0);
    int *d_scanArray;
    d_scanArray = hillisSteeleScanHost(d_predicateArrry,numberOfElements);
    int *d_scatteredArray;
    hipMalloc((void**)&d_scatteredArray,sizeof(int)*numberOfElements);
    //cout<<"offset = "<<offset<<"\n";
    scatter<<<1600,500>>>(d_array,d_scanArray,d_predicateArrry,d_scatteredArray, numberOfElements,0);
    hipMemcpy(&offset,d_scanArray+numberOfElements-1,sizeof(int),hipMemcpyDeviceToHost);
    predicateDevice<<<1600,500>>>(d_array,d_predicateArrry,numberOfElements,bit,1);
    d_scanArray = hillisSteeleScanHost(d_predicateArrry,numberOfElements);
    scatter<<<1600,500>>>(d_array,d_scanArray,d_predicateArrry,d_scatteredArray, numberOfElements,offset);
    return d_scatteredArray;
}
int *compact2(int *d_array,int numberOfElements,int bit)
{
    int offset;
    int *d_predicateArrry;
    hipMalloc((void**)&d_predicateArrry,sizeof(int)*numberOfElements);
    predicateDevice<<<1600,500>>>(d_array,d_predicateArrry,numberOfElements,bit,1);
    int *d_scanArray;
    d_scanArray = hillisSteeleScanHost(d_predicateArrry,numberOfElements);
    int *d_scatteredArray;
    hipMalloc((void**)&d_scatteredArray,sizeof(int)*numberOfElements);
    //cout<<"offset = "<<offset<<"\n";
    scatter<<<1600,500>>>(d_array,d_scanArray,d_predicateArrry,d_scatteredArray, numberOfElements,0);
    hipMemcpy(&offset,d_scanArray+numberOfElements-1,sizeof(int),hipMemcpyDeviceToHost);
    predicateDevice<<<1600,500>>>(d_array,d_predicateArrry,numberOfElements,bit,0);
    d_scanArray = hillisSteeleScanHost(d_predicateArrry,numberOfElements);
    scatter<<<1600,500>>>(d_array,d_scanArray,d_predicateArrry,d_scatteredArray, numberOfElements,offset);
    return d_scatteredArray;
}
int offset;
int *positivenegativesplit(int *d_array,int numberOfElements,int bit,int bitset)
{   
    int *d_predicateArrry;
    hipMalloc((void**)&d_predicateArrry,sizeof(int)*numberOfElements);
    predicateDevice<<<1600,500>>>(d_array,d_predicateArrry,numberOfElements,bit,bitset);
    int *d_scanArray;
    d_scanArray = hillisSteeleScanHost(d_predicateArrry,numberOfElements);
    int *d_scatteredArray;
    hipMemcpy(&offset,d_scanArray+numberOfElements-1,sizeof(int),hipMemcpyDeviceToHost);
    hipMalloc((void**)&d_scatteredArray,sizeof(int)*offset);
    //cout<<"offset = "<<offset<<"\n";
    scatter<<<1600,500>>>(d_array,d_scanArray,d_predicateArrry,d_scatteredArray, numberOfElements,0);
    return d_scatteredArray;
}
int * radixSort(int *d_array , int numberOfElements)
{
    int bit;
    int *d_negativeArray = positivenegativesplit(d_array,numberOfElements,1L<<31,1);
    for(int i=0;i<sizeof(int)*8;i++)
    {
        bit = 1<<i;
        d_negativeArray = compact2(d_negativeArray,offset,bit);
    }
    int *d_postiveArray = positivenegativesplit(d_array,numberOfElements,1L<<31,0);
    for(int i=0;i<sizeof(int)*8;i++)
    {
        bit = 1<<i;
        d_postiveArray = compact(d_postiveArray,offset,bit);
    }
    hipMemcpy(d_array,d_negativeArray,sizeof(int)*(numberOfElements-offset),hipMemcpyDeviceToDevice);
    hipMemcpy(d_array+(numberOfElements-offset),d_postiveArray,sizeof(int)*offset,hipMemcpyDeviceToDevice);
    return d_array;
}
int main()
{
    cout<<"enter the number of elements \n";
    int numberOfElements;
    cin>>numberOfElements;
    int *h_array  = new int[numberOfElements];
    //class random a(h_array,numberOfElements);
    for(int i=0;i<numberOfElements;i++)
    {
        cin>>h_array[i];
    }
    int *d_array;
    hipMalloc((void**)&d_array ,sizeof(int)*numberOfElements);
    hipMemcpy(d_array,h_array,sizeof(int)*numberOfElements,hipMemcpyHostToDevice);
    d_array = radixSort(d_array, numberOfElements);
    hipMemcpy(h_array,d_array,sizeof(int)*numberOfElements,hipMemcpyDeviceToHost);
    for(int i=0;i<numberOfElements;i++)
    {
        cout<<h_array[i]<<"\n";
    }
}
