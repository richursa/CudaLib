
#include <hip/hip_runtime.h>
#include<iostream>
#include<stdio.h>
using namespace std;
__global__ void predicateDevice(int *d_array , int *d_predicateArrry , int d_numberOfElements,int bit,int bitset)
{
    int index = threadIdx.x + blockIdx.x*blockDim.x;
    if(index < d_numberOfElements)
    {
        if(bitset == 0)
        {
            if((d_array[index] & bit) == 0)
             {
                d_predicateArrry[index] = 1;
             }
             else
             {
                d_predicateArrry[index] = 0;
             }
        }
        else
        {
            if((d_array[index] & bit) != 0)
            {
                d_predicateArrry[index] = 1;
            }
            else
            {
                d_predicateArrry[index] = 0;
            }
        }
    }
}
__global__ void scatter(int *d_array , int *d_scanArray , int *d_predicateArrry,int * d_scatteredArray ,int d_numberOfElements,int offset)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if(index < d_numberOfElements)
    {
        if(d_predicateArrry[index] == 1)
        {
            d_scatteredArray[d_scanArray[index] - 1 +offset ] = d_array[index];
        
        }
    }
}

__global__ void hillisSteeleScanDevice(int *d_array , int numberOfElements, int *d_tmpArray,int moveIndex)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    if(index > numberOfElements)
    {
        return;
    }
    d_tmpArray[index] = d_array[index];
    if(index - moveIndex >=0)
    {
        
        d_tmpArray[index] = d_tmpArray[index] +d_array[index - moveIndex];
    }
}
int* hillisSteeleScanHost(int *d_scanArray,int numberOfElements)
{
    int *d_tmpArray;
    hipMalloc(&d_tmpArray,sizeof(int)*numberOfElements);
    int j,k=0;
    for(j=1;j<numberOfElements;j= j*2,k++)
    {
        if(k%2 == 0)
        {
            hillisSteeleScanDevice<<<100,256>>>(d_scanArray,numberOfElements,d_tmpArray, j);
        }
        else
        {
            hillisSteeleScanDevice<<<100,256>>>(d_tmpArray,numberOfElements,d_scanArray, j);
        }
    } 
    hipDeviceSynchronize();
    if(k%2 == 0)
    {
        return d_scanArray;
    }
    else
    {
        return d_tmpArray;
    }
}

__global__ void getPos(int *d_scanArray , int d_numberOfElements,int *d_lastPos)
{
    *d_lastPos = d_scanArray[d_numberOfElements -1];
}
void radix(int *h_array , int numberOfElements,int numberOfThreads ,int numberOfBlocks)
{
    int *d_array ;
    hipMalloc((void**)&d_array,sizeof(int)*numberOfElements);
    hipMemcpy(d_array,h_array,sizeof(int)*numberOfElements,hipMemcpyHostToDevice);
    int *d_predicateArrry;
    hipMalloc((void**)&d_predicateArrry , sizeof(int)*numberOfElements);
    int *d_scanArray;
    hipMalloc((void**)&d_scanArray,sizeof(int)*numberOfElements);
    int *d_scatteredArray;
    hipMalloc((void**)&d_scatteredArray,sizeof(int)*numberOfElements);
    int *d_lastPos;
    hipMalloc ((void**)&d_lastPos,sizeof(int));
    int *h_lastPos = new int[1];
    for(int i=0;i<8*sizeof(int);i++)
    {
        predicateDevice<<<numberOfBlocks,numberOfThreads>>>(d_array,d_predicateArrry,numberOfElements,1<<(i),0);
        hipMemcpy(d_scanArray,d_predicateArrry,sizeof(int)*numberOfElements,hipMemcpyDeviceToDevice);
        d_scanArray = hillisSteeleScanHost(d_scanArray,numberOfElements);
        scatter<<<numberOfBlocks,numberOfElements>>>(d_array,d_scanArray,d_predicateArrry,d_scatteredArray,numberOfElements,0);
        getPos<<<1,1>>>(d_scanArray,numberOfElements,d_lastPos);
        predicateDevice<<<numberOfBlocks,numberOfThreads>>>(d_array,d_predicateArrry,numberOfElements,1<<(i),1);
        hipMemcpy(d_scanArray,d_predicateArrry,sizeof(int)*numberOfElements,hipMemcpyDeviceToDevice);
        d_scanArray = hillisSteeleScanHost(d_scanArray,numberOfElements);
        hipMemcpy(h_lastPos,d_lastPos,sizeof(int),hipMemcpyDeviceToHost);
        scatter<<<numberOfBlocks,numberOfThreads>>>(d_array,d_scanArray,d_predicateArrry,d_scatteredArray,numberOfElements,(*h_lastPos));
        hipMemcpy(d_array,d_scatteredArray,sizeof(int)*numberOfElements,hipMemcpyDeviceToDevice);

    }
    hipMemcpy(h_array,d_array,sizeof(int)*numberOfElements,hipMemcpyDeviceToHost);
}


int main()
{
    cout<<"enter the numbre of element";
    int numberOfElements;
    cin>>numberOfElements;
    int *h_array = new int[numberOfElements];
    //class random a(h_array ,numberOfElements);
    for(int i=numberOfElements-1,k=0;i>=0;i--,k++)
    {
        h_array[k] = i;
    }
    radix(h_array,numberOfElements,256,100);
    hipDeviceSynchronize();
    for(int i=0;i<numberOfElements;i++)
    {
        cout<<h_array[i]<<"\n";
    }
    
}