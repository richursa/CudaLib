#include<stdio.h>
#include<hip/hip_runtime.h>
#include<iostream>
#include<fstream>
#include<chrono>


using namespace std;
__global__ void parallelReduction(int *d_array , int numberOfElements, int elementsPerThread,int numberOfThreadsPerBlock,int numberOfBlocks,int *d_global)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x ;
    int sum = 0;

    int j=0;
    for(int i=index;i<numberOfElements;i = i+(numberOfBlocks*numberOfThreadsPerBlock))
    {
        sum = sum + d_array[i];
        j++;
    }
    d_global[index] = sum;
}

void parallelReduceHost(int *h_array ,int *d_array ,int numberOfElements,int elementsPerThread , int numberOfThreadsPerBlock , int numberOfBlocks)
{
    int *d_global;
    hipMalloc(&d_global, sizeof(int)*numberOfBlocks*numberOfThreadsPerBlock);
    parallelReduction<<<numberOfBlocks,numberOfThreadsPerBlock>>>(d_array,numberOfElements,elementsPerThread,numberOfThreadsPerBlock,numberOfBlocks,d_global);
    int *d_global1;
    hipMalloc(&d_global1,sizeof(int)*numberOfThreadsPerBlock*numberOfBlocks);
    parallelReduction<<<numberOfBlocks,numberOfThreadsPerBlock>>>(d_global,2560*64,elementsPerThread,64,80,d_global1);
    int *h_global = new int[64*80];
    hipMemcpy(h_global,d_global1,sizeof(int)*64*80,hipMemcpyDeviceToHost);
    int sum=0;
    for(int i=0;i<64*80;i++)
    {
            sum =sum+h_global[i];
    }
    cout<<sum;
}
int main()
{
   
    int numberOfElements;
    ifstream inFile;
    inFile.open("random");
    int x;
    int i=0;
    inFile >>x ;
    numberOfElements = x;
   int *h_array = new int[numberOfElements];
    while(inFile >> x)
    {
        h_array[i] = x;
        i++;
    }
    int *d_array;
    hipMalloc(&d_array , sizeof(int)*numberOfElements);
    hipMemcpy(d_array, h_array , sizeof(int)*numberOfElements, hipMemcpyHostToDevice);
    //serialReduceHost(h_array, d_array ,numberOfElements);
    int elementsPerThread, numberOfBlocks , numberOfThreadsPerBlock;
   elementsPerThread = 0 ;
   numberOfThreadsPerBlock = 64;
   numberOfBlocks =2560;
    parallelReduceHost(h_array,d_array,numberOfElements,elementsPerThread,numberOfThreadsPerBlock ,numberOfBlocks);  


}   
