#include "hip/hip_runtime.h"
#include<iostream>
#include<stdio.h>
#include "random.h"
#include<conio.h>
using namespace std;
int h_sizeOfCompactedArray;
__global__ void scatter(int *d_array , int *d_predicateArray, int *d_scanArray,int *d_compactedArray, int d_numberOfElements)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index < d_numberOfElements)
    {
        if(d_predicateArray[index]==1)
        {
            d_compactedArray[d_scanArray[index]-1] = d_array[index];
        
        }
    }
}

__global__ void predicate(int *d_array, int d_numberOfElements,int *d_predicateArray)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index <d_numberOfElements)
    {
        if(d_array[index]%32== 0)
        {
            d_predicateArray[index] =1;
        }
        else
        {
            d_predicateArray[index]  = 0;
        }
    }
}

__global__ void hillisSteeleScanDevice(int *d_predicateArray , int d_numberOfElements ,int *d_tmpArray,int d_offset)
{
    int index = blockIdx.x * blockDim.x +  threadIdx.x;
    if(index < d_numberOfElements)
    {
        d_tmpArray[index] = d_predicateArray[index];
        if(index - d_offset >= 0)
            {
            
             d_tmpArray[index] = d_predicateArray[index] + d_predicateArray[index-d_offset];
            }
    }
}
void hillisSteeleScanHost(int *d_predicateArray,int h_numberOfElements)
{
    int k=0;
    int *d_tmpArray;
    hipMalloc(&d_tmpArray,sizeof(int)*h_numberOfElements);
    for(int j=1;j<h_numberOfElements;j= j*2,k++)
    {
        if(k%2==0)
        {
            hillisSteeleScanDevice<<<1600,500>>>(d_predicateArray,h_numberOfElements,d_tmpArray,j);
        }
        else
        {
            hillisSteeleScanDevice<<<1600,500>>>(d_tmpArray,h_numberOfElements,d_predicateArray,j);
        }
        
    }
    if(k%2==0)
    {
        
    }
    else
    {
        d_predicateArray = d_tmpArray;
    }
}

void normalPredicarte(int *h_array, int h_numberOfElements)
{
    cout<<"\ncpu muwth\n";
    int j=0;
    for(int i=0;i<h_numberOfElements;i++)
    {
        if(h_array[i]%32 == 0)
        {
            cout<<h_array[i]<<"\n";
            j++;
        }
    }
    
        cout<<"\n size of compact cpu "<<j<<"\n";
        h_sizeOfCompactedArray = j;
    cout<<"gpu freak\n";
}
int main()
{
    cout<<"enter the number of elements";
    int h_numberOfElements;
    cin>>h_numberOfElements;
    int *h_array = new int[h_numberOfElements];
    class random a(h_array,h_numberOfElements);
    normalPredicarte(h_array,h_numberOfElements);
    int *d_array;
    hipMalloc(&d_array,sizeof(int)*h_numberOfElements);
    hipMemcpy(d_array,h_array,sizeof(int)*h_numberOfElements,hipMemcpyHostToDevice);
    int *d_predicateArray;
    hipMalloc(&d_predicateArray,sizeof(int)*h_numberOfElements);
    int *d_scanArray;
    hipMalloc(&d_scanArray,sizeof(int)*h_numberOfElements);
    predicate<<<1600 ,500>>>(d_array,h_numberOfElements,d_predicateArray);
    hipMemcpy(d_scanArray,d_predicateArray,sizeof(int)*h_numberOfElements,hipMemcpyDeviceToDevice);
    hillisSteeleScanHost(d_scanArray,h_numberOfElements);
    int *d_compactedArray;
    //int h_sizeOfCompactedArray;
   // hipMemcpy(&h_sizeOfCompactedArray,&d_scanArray[h_numberOfElements-2],sizeof(int),hipMemcpyDeviceToHost);
    cout<<"\nsize of compacted array "<<h_sizeOfCompactedArray<<"\n";
    hipMalloc(&d_compactedArray,sizeof(int)*h_sizeOfCompactedArray);
    scatter<<<1600,500>>>(d_array,d_predicateArray,d_scanArray,d_compactedArray,h_numberOfElements);
    int *h_compactedArray = new int[h_sizeOfCompactedArray];
    hipMemcpy(h_compactedArray,d_compactedArray,sizeof(int)*h_sizeOfCompactedArray,hipMemcpyDeviceToHost);
    for(int i=0;i<h_sizeOfCompactedArray;i++)
        {
         cout<<h_compactedArray[i]<<"\n";
        }
    cout<<"scanarray is \n";
    fflush(stdin);
    hipDeviceSynchronize();
    hipMemcpy(h_array,d_scanArray,sizeof(int)*h_numberOfElements,hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    getch();

}
/*int main()
{
    int d_numberOfElements;
    cin>>d_numberOfElements;
    int *h_array = new int[d_numberOfElements];
    for(int i=0;i<d_numberOfElements;i++)
    {
        h_array[i]  = i;
    }
    int *d_array;
    hipMalloc(&d_array,sizeof(int)*d_numberOfElements);
    hipMemcpy(d_array,h_array,sizeof(int)*d_numberOfElements,hipMemcpyHostToDevice);
    hillisSteeleScanHost(d_array,d_numberOfElements);
    hipMemcpy(h_array,d_array,sizeof(int)*d_numberOfElements,hipMemcpyDeviceToHost);
    for(int i=0;i<d_numberOfElements;i++)
    {
        cout<<h_array[i]<<"\n";
    }
    getch();
}*/